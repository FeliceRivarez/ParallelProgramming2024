// g++ ./openMPRow.cpp -fopenmp -o ./openMPRow.o -std=c++11
//nvcc -o RowCUDA RowCUDA.cu -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Tools\MSVC\14.39.33519\bin\Hostx64\x64\cl.exe"
#include <iostream>
#include <queue>
#include <fstream>
#include <chrono>
#include <omp.h>
#include "hip/hip_runtime.h"

using namespace std;
using namespace std::chrono;
const int NUM_THREADS = 6;

__global__ void calculate_mt(char *a, char *b, int m, int n, int **mt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m + 1 && i1 < n + 1)
    {
        if (a[i - 1] == b[i1 - 1])
        {
            mt[i][i1] = i1;
        }
        else
        {
            mt[i][i1] = 0;
        }
    }
}

__global__ void calculate_dp(char *a, char *b, int m, int n, int **dp, int **mt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int i1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m + 1 && i1 < n + 1)
    {
        if (a[i - 1] == b[i1 - 1])
        {
            dp[i][i1] = dp[i - 1][i1 - 1];
        }
        else
        {
            int nearest = mt[i][i1];
            if (nearest == 0)
            {
                dp[i][i1] = min(min(dp[i - 1][i1], dp[i - 1][i1 - 1]), dp[i - 1][nearest] + i1 - nearest - 1) + 1;
            }
            else
            {
                dp[i][i1] = min(min(dp[i - 1][i1], dp[i - 1][i1 - 1]), dp[i - 1][nearest - 1] + i1 - nearest - 1) + 1;
            }
        }
    }
}

int main()
{
    ofstream result("result.txt");
    fstream testset("testset.txt");
    string a = "114514";
    string b = "1919810";
    int test = 10;
    long long total = 0;

    while (testset >> a)
    {
        testset >> b;
        // a="114514";
        // b="1919810";
        int distance = 0;
        for (int k = 0; k < test; k += 1)
        {
            int m = a.length();
            int n = b.length();
            int **dp = new int *[m + 1];
            for (int i = 0; i < m + 1; i += 1)
            {
                dp[i] = new int[n + 1];
            }
            for (int i = 0; i < m + 1; i += 1)
            {
                dp[i][0] = i;
            }
            for (int i = 0; i < n + 1; i += 1)
            {
                dp[0][i] = i;
            }
            auto start = system_clock::now();
            int **mt = new int *[m + 1];
            for (int i = 0; i < m + 1; i += 1)
            {
                mt[i] = new int[n + 1];
            }

            for (int i = 0; i < m + 1; i += 1)
            {
                mt[i][0] = 0;
            }
            for (int i = 0; i < n + 1; i += 1)
            {
                mt[0][i] = 0;
            }
            int i = 0;
            int i1 = 1;
            cout << "here" << endl;
            // 分配设备内存
            int *dp_device, *mt_device;
            hipMalloc(&dp_device, (m + 1) * (n + 1) * sizeof(int)); // 分配 (m+1) * (n+1) 个 int 的空间
            hipMalloc(&mt_device, (m + 1) * (n + 1) * sizeof(int)); // 分配 (m+1) * (n+1) 个 int 的空间

            // 循环中为每个指针分配足够的内存空间
            int **dp_host = new int *[m + 1]; // 分配 m+1 个 int* 指针的空间
            int **mt_host = new int *[m + 1]; // 分配 m+1 个 int* 指针的空间
            for (int i = 0; i < m + 1; i++)
            {
                dp_host[i] = new int[n + 1]; // 为每个指针分配 (n+1) 个 int 的空间
                mt_host[i] = new int[n + 1]; // 为每个指针分配 (n+1) 个 int 的空间
            }

            // 将数据传输到设备
            char *a_device, *b_device;
            hipMalloc(&a_device, a.length() * sizeof(char));
            hipMalloc(&b_device, b.length() * sizeof(char));
            hipMemcpy(a_device, a.c_str(), a.length() * sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(b_device, b.c_str(), b.length() * sizeof(char), hipMemcpyHostToDevice);

            // 启动内核
            // dim3 threadsPerBlock(16, 16);
            // dim3 blocksPerGrid((m + threadsPerBlock.x - 1) / threadsPerBlock.x, (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
            // calculate_mt<<<blocksPerGrid, threadsPerBlock>>>(a_device, b_device, m, n, mt_device);
            // calculate_dp<<<blocksPerGrid, threadsPerBlock>>>(a_device, b_device, m, n, dp_device, mt_device);

            // 将结果传输回主机
            // cudaMemcpy(dp, dp_device, (m + 1) * sizeof(int *), cudaMemcpyDeviceToHost);
            // for (int i = 0; i < m + 1; i++)
            // {
            //     cudaMemcpy(dp[i], dp_device[i], (n + 1) * sizeof(int), cudaMemcpyDeviceToHost);
            // }
            cout << "here" << endl;
            auto duration = duration_cast<microseconds>(system_clock::now() - start);
            cout << duration.count() << endl;
            total += duration.count();
            cout << dp[m][n] << endl;
            distance = dp[m][n];

            for (int i = 0; i < m + 1; i += 1)
            {
                delete dp[i];
            }
            for (int i = 0; i < m + 1; i += 1)
            {
                delete mt[i];
            }
            for (int i = 0; i < m + 1; i++)
            {
                //cudaFree(dp_device[i]);
                //cudaFree(mt_device[i]);
            }
            hipFree(dp_device);
            hipFree(mt_device);
            hipFree(a_device);
            hipFree(b_device);
        }
        result << distance << " " << total / 10 << endl;
        total = 0;
    }
    // system("pause");
}
