#include <iostream>
#include <hip/hip_runtime.h> // Include the CUDA runtime library



// CUDA kernel function for vector addition
__global__ void vector_add(float* out, float* a, float* b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        out[index] = a[index] + b[index];
    }
}

int main() {
    const int N = 114514; // 1 million elements
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_a = (float*)malloc(size);
    float* h_b = (float*)malloc(size);
    float* h_out = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    // Allocate device memory
    float* d_a, * d_b, * d_out;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add  <<<blocksPerGrid, threadsPerBlock>>>(d_out, d_a, d_b, N);

    // Copy result back to host
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // Verify result
    float sum = 0.0f;
    for (int i = 0; i < N; ++i) {
        sum += h_out[i];
    }
    std::cout << "Sum of array elements: " << sum << std::endl;
 
    // Clean up
    free(h_a);
    free(h_b);
    free(h_out);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return 0;
}
