//nvcc -o CUDAopenMP CUDAopenMP.cu -ccbin "C:\Program Files\Microsoft Visual Studio\2022\Community\VC\Tools\MSVC\14.39.33519\bin\Hostx64\x64\cl.exe" -Xcompiler " -openmp" 
#include <iostream>
#include <queue>
#include <fstream>
#include <chrono>
// #include <omp.h>
using namespace std;
using namespace std::chrono;
const int NUM_THREADS = 6;
#include "hip/hip_runtime.h"


__global__ void levenshtein_kernel(int *dp, int *dp_prev, int *mt_row, int m, int n)
{
    int i1 = blockIdx.x * blockDim.x + threadIdx.x;

    if (i1 < n + 1)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
        int nearest = mt_row[i1];
        if (nearest == 0)
        {
            dp[i1] = min(dp_prev[i1], min(dp_prev[i1 - 1], dp_prev[nearest] + i1 - nearest - 1)) + 1;
        }
        else
        {
            dp[i1] = min(dp_prev[i1], min(dp_prev[i1 - 1], dp_prev[nearest - 1] + i1 - nearest - 1)) + 1;
        }
    }
}

int main()
{
    ofstream result("result.txt");
    fstream testset("testset.txt");
    string a = "114514";
    string b = "1919810";
    int test = 10;
    long long total = 0;

    while (testset >> a)
    {
        testset >> b;
        // a = "114514";
        // b = "1919810";
        int distance = 0;
        for (int k = 0; k < test; k += 1)
        {
            int m = a.length();
            int n = b.length();
            int *dp = new int[n + 1];
            int *dp_prev = new int[n + 1];

            // int **dp=new int*[m+1];
            // for(int i=0; i<m+1; i+=1)
            // {
            //     dp[i]=new int[n+1];
            // }
            // for(int i=0; i<m+1; i+=1)
            // {
            //     dp[i][0]=i;
            // }
            for (int i = 0; i < n + 1; i += 1)
            {
                dp_prev[i] = i;
            }
            dp[0] = 1;
            auto start = system_clock::now();
            int **mt = new int *[m + 1];
            for (int i = 0; i < m + 1; i += 1)
            {
                mt[i] = new int[n + 1];
            }

            for (int i = 0; i < m + 1; i += 1)
            {
                mt[i][0] = 0;
            }
            for (int i = 0; i < n + 1; i += 1)
            {
                mt[0][i] = 0;
            }
            cout << "!" << endl;
            int i = 0;
            int i1 = 1;
#pragma omp parallel num_threads(NUM_THREADS), private(i, i1), default(shared)
#pragma omp for
            for (int i = 1; i < m + 1; i += 1)
            {
                int nearest = 0;
                for (int i1 = 1; i1 < n + 1; i1 += 1)
                {
                    if (a[i - 1] == b[i1 - 1])
                    {
                        nearest = i1;
                    }
                    mt[i][i1] = nearest;
                }
            }
            auto duration1=duration_cast<microseconds>(system_clock::now() - start);
            int *d_dp, *d_dp_prev, *d_mt_row;
            hipMalloc((void **)&d_dp, (n + 1) * sizeof(int));
            hipMalloc((void **)&d_dp_prev, (n + 1) * sizeof(int));
            hipMalloc((void **)&d_mt_row, (n + 1) * sizeof(int));

            // 初始化dp和dp_prev
            int *h_dp = new int[n + 1];
            int *h_dp_prev = new int[n + 1];
            for (int i1 = 0; i1 < n + 1; i1++)
            {
                h_dp_prev[i1] = i1;
            }
            hipMemcpy(d_dp_prev, h_dp_prev, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
            int memcpy=0;
            int vacancy=0;
            for (int i = 1; i < m + 1; i += 1)
            {
                auto duration2=duration_cast<microseconds>(system_clock::now() - start);
                // 将当前行的mt数组传输到GPU
                hipMemcpy(d_mt_row, mt[i], (n + 1) * sizeof(int), hipMemcpyHostToDevice);
                auto duration3=duration_cast<microseconds>(system_clock::now() - start);
                memcpy+=duration3.count()-duration2.count();

                // 调用CUDA核函数
                dim3 blockDim(128, 1);                              // 根据需要调整线程块大小
                dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1); // 根据需要调整网格大小
                levenshtein_kernel<<<gridDim, blockDim>>>(d_dp, d_dp_prev, d_mt_row, m, n);

                // 将结果传输回主机
                hipMemcpy(h_dp, d_dp, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
                auto duration4=duration_cast<microseconds>(system_clock::now() - start);
                vacancy+=duration4.count()-duration3.count();
                // 交换dp和dp_prev
                int *temp = h_dp;
                h_dp = h_dp_prev;
                h_dp_prev = temp;
                h_dp_prev[0] = i; // 设置dp[0]
                h_dp[0] = i + 1;
                // for (int i1 = 0; i1 < n + 1; i1 += 1)
                // {
                //     cout << h_dp_prev[i1] << " ";
                // }
                // cout << endl;
                // 将新的dp_prev拷贝回GPU
                // if (i == m)
                // {
                //     cout << h_dp_prev[n] << endl;
                // }
                distance=h_dp_prev[n];
                hipMemcpy(d_dp_prev, h_dp_prev, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
            }

            // 计算距离
            //cudaMemcpy(&distance, h_dp_prev + n, sizeof(int), cudaMemcpyDeviceToHost);

            auto duration = duration_cast<microseconds>(system_clock::now() - start);
            cout << duration.count() << endl;
            cout<<"mt time:"<<duration1.count()<<endl;
            cout<<"memcpy:"<<memcpy<<endl;
            cout<<"vacancy:"<<vacancy<<endl;
            total += duration.count();
            cout << distance << endl;
            memcpy=0;
            vacancy=0;
            // 释放内存
            delete[] h_dp;
            delete[] h_dp_prev;
            for (int i = 0; i < m + 1; i++)
            {
                delete[] mt[i];
            }
            delete[] mt;
            hipFree(d_dp);
            hipFree(d_dp_prev);
            hipFree(d_mt_row);
        }
        result << distance << " " << total / 10 << endl;
        total = 0;
    }
    // system("pause");
}
